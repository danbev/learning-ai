#include <hip/hip_runtime.h>

#include <stdio.h>

template<typename T> __global__ void add_arrays(const T* a, const T* b, T* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 6;
    int size = N * sizeof(int);

    int* h_a = new int[N];
    int* h_b = new int[N];
    int* h_c = new int[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    printf("array a:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_a[i]);
    }
    printf("\n");
    printf("array b:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_b[i]);
    }
    printf("\n");

    int* d_a;
    int* d_b;
    int* d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    dim3 blocks(2);  // blocks per grid
    dim3 threads(3); // threads per block
    add_arrays<<<blocks, threads>>>(d_a, d_b, d_c, N);

    hipDeviceSynchronize();

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "array_add kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    // Copy the array that the device has computed back to the host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    printf("Added on GPU:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_c[i]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    return 0;
}

